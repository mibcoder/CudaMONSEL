
#include <hip/hip_runtime.h>
//#include "ElementTest.cuh"
//#include "..\EPQLibrary\Element.cuh"
//#include "..\EPQLibrary\FromSI.cuh"
//
//#include <stdio.h>
//#include <math.h>
//
//namespace ElementTest
//{
//   __device__ void assertEquals(double src, double target, double delta)
//   {
//      bool b = fabs(src - target) < delta;
//      if (!b) {
//         printf("ElementTest::assertEquals: values are different: %lf, %lf\n", src, target);
//      }
//   }
//
//   __device__ void assertEquals(int src, int target)
//   {
//      if (src != target) {
//         printf("ElementTest::assertEquals: values are different: %d, %d\n", src, target);
//      }
//   }
//
//   __device__ void assertTrue(bool expr)
//   {
//      if (!expr) {
//         printf("ElementTest::assertTrue: expr is not true\n");
//      }
//   }
//
//   __device__ ElementTest::ElementTest()
//   {
//   }
//
//   __device__ void ElementTest::testOne()
//   {
//      Element::Element elm = Element::byAtomicNumber(Element::elmTi);
//      //assertEquals(FromSI::eV(elm.meanIonizationPotential()), 247.24, 1.0);
//      assertEquals(elm.getAtomicWeight(), 47.9, 1.0e-1);
//      assertEquals(elm.getAtomicNumber(), 22);
//      assertTrue(elm.compareTo(Element::byAtomicNumber(Element::elmH)) > 0);
//      assertTrue(elm.compareTo(Element::byAtomicNumber(Element::elmFe)) < 0);
//      assertTrue(elm.compareTo(Element::byAtomicNumber(Element::elmTi)) == 0);
//      assertTrue(elm.equals(Element::byName("Ti")));
//   }
//}
